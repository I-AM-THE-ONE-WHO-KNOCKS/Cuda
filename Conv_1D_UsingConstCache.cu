#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <iostream>

#define MASK_LEN 8

/*as mask is never changing we can define a constant memory on the device side so that
 we do not have to copu again and again and loading from const cache is much much faster that 
 loading from d-ram.
*/
__constant__ int mask[MASK_LEN];

__global__ void conv_1d(int* a, int* c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	//cal the radius of the mask(mid point)
	int r = MASK_LEN / 2;
	//cal the start point of for the element
	int start = id - r;
	int temp = 0;
	for (int j = 0; j < MASK_LEN; j++)
	{
		if ((start + j >= 0) && (start + j < n))
		{
			temp += a[start + j] * mask[j];
		}
	}
	c[id] = temp;
}

// Initialize
void Array_init(int* a, int n, int div) {
	for (int i = 0; i < n; i++) {
			a[i] = rand() % div;
		}
}

void check_answer(int* a, int* b, int* c, int n, int m) {
	int radius = m / 2;
	int temp;
	int start;
	for (int i = 0; i < n; i++)
	{
		start = i - radius;
		temp = 0;
		for (int j = 0; j < m; j++)
		{
			if ((start + j >= 0) && (start + j < n))
			{
				temp += a[start + j] * b[j];
			}
		}
		assert(temp == c[i]);
	}
}

int main() {
	
	// number of elements in result array
	int n = 1 << 16;
	
	int n_bytes = n * sizeof(int);

	//num of elemets in mask
	int m = 8;

	int m_bytes = m * sizeof(int);

	//allocate the array
	int* h_arr = new int[n];

	Array_init(h_arr, n, 100);

	//allocate the mask and intialize it
	int* h_mask = new int[m];
	Array_init(h_mask, m, 10);

	//allocate space for result
	int* h_result = new int[n];

	//allocate space on device memory
	int* d_arr, * d_res;
	hipMalloc(&d_arr, n_bytes);
	hipMalloc(&d_res, n_bytes);

	hipMemcpy(d_arr, h_arr, n_bytes, hipMemcpyHostToDevice);
	//special function to copy to a symbol
	hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, m_bytes);

	int threads = 256;

	int grid = (n + threads - 1) / threads;

	conv_1d <<<grid, threads>>> (d_arr, d_res, n);

	hipMemcpy(h_result, d_res, n_bytes, hipMemcpyDeviceToHost);

	check_answer(h_arr, h_mask, h_result, n, m);

	free(h_result);
	free(h_mask);
	free(h_arr);

	hipFree(d_arr);
	hipFree(d_res);

	printf("COMPLETED SUCCESFULLY\n");

	return 0;
}